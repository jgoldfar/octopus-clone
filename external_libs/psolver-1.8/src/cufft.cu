#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime_api.h"
#include "config.h"
 
#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8


static const char *_cublasGetErrorString(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";
#if CUDA_VERSION >= 6500
        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";
#endif
    }
    return "<unknown>";
}

static const char *_cufftGetErrorString(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
    }
    return "<unknown>";
}


extern hipStream_t stream1;
extern hipblasHandle_t handle1;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define cufftErrchk(ans) { __cufftAssert((ans), __FILE__, __LINE__); }

inline void __cufftAssert(hipfftResult code, const char *file, const int line, bool abort=true)
{
   if(code != HIPFFT_SUCCESS) 
   {
      fprintf(stderr, "cufftAssert : %s %s %d.\n",
      _cufftGetErrorString(code), file, line);
      if (abort) exit(-1);
   }
}

#define cublasErrchk(ans) { __cublasAssert((ans), __FILE__, __LINE__); }

inline void __cublasAssert(hipblasStatus_t code, const char *file, const int line, bool abort=true)
{
   if(code !=HIPBLAS_STATUS_SUCCESS) 
   {
      fprintf(stderr, "cublasAssert : %s %s %d.\n",
      _cublasGetErrorString(code), file, line);
      if (abort) exit(-1);
   }
}


// create stream for kernel
extern "C" void FC_FUNC(cudacreatestream, CUDACREATESTREAM) (int* ierr) {
  *ierr = hipStreamCreate(&stream1);
}

extern "C" void FC_FUNC(cudadestroystream, CUDADESTROYSTREAM) (int* ierr) {
  *ierr = hipStreamDestroy(stream1);
}

// create stream for kernel
extern "C" void FC_FUNC(cudacreatecublashandle, CUDACREATECUBLASHANDLE) () {
  cublasErrchk(hipblasCreate(&handle1));
}

extern "C" void FC_FUNC(cudadestroycublashandle, CUDADESTROYCUBLASHANDLE) () {
  cublasErrchk(hipblasDestroy(handle1));
}


extern "C" void FC_FUNC(cufftdestroy, CUFFTDESTROY) (hipfftHandle *plan) {
  hipfftDestroy(*plan);
}

// set device memory
extern "C" void FC_FUNC_(send_and_pad_data, SEND_AND_PAD_DATA)(Real* h_data, Real **d_data, int* m1, int* m2, int*m3, int* md1, int*md2, int* md3){

hipMemsetAsync(*d_data, 0, *md1**md2**md3*sizeof(Real),stream1);
hipMemcpy3DParms cpyParms = {0};

cpyParms.srcPtr = make_hipPitchedPtr(h_data, ((size_t)*m1)*sizeof(Real), ((size_t)*m2), ((size_t)*m3));

cpyParms.dstPtr = make_hipPitchedPtr(*d_data, ((size_t)*md1)*sizeof(Real), ((size_t)*md2), ((size_t)*md3));

cpyParms.extent = make_hipExtent( ((size_t)*m1)*sizeof(Real),  ((size_t)*m3),  ((size_t)*m2));;
cpyParms.kind = hipMemcpyHostToDevice;

hipError_t status = hipMemcpy3DAsync(&cpyParms,stream1);

if(status != hipSuccess){fprintf(stderr, "%s\n", hipGetErrorString(status));}

}

// set device memory
extern "C" void FC_FUNC_(pad_data, PAD_DATA)(Real** h_data, Real **d_data, int* m1, int* m2, int*m3, int* md1, int*md2, int* md3){

hipMemsetAsync(*d_data, 0, *md1**md2**md3*sizeof(Real),stream1);
hipMemcpy3DParms cpyParms = {0};

cpyParms.srcPtr = make_hipPitchedPtr(*h_data, ((size_t)*m1)*sizeof(Real), ((size_t)*m2), ((size_t)*m3));

cpyParms.dstPtr = make_hipPitchedPtr(*d_data, ((size_t)*md1)*sizeof(Real), ((size_t)*md2), ((size_t)*md3));

cpyParms.extent = make_hipExtent( ((size_t)*m1)*sizeof(Real),  ((size_t)*m3),  ((size_t)*m2));;
cpyParms.kind = hipMemcpyDeviceToDevice;

hipError_t status = hipMemcpy3DAsync(&cpyParms,stream1);

if(status != hipSuccess){fprintf(stderr, "%s\n", hipGetErrorString(status));}

}


// set device memory
extern "C" void FC_FUNC_(unpad_data, UNPAD_DATA)(Real** h_data, Real **d_data, int* m1, int* m2, int*m3, int* md1, int*md2, int* md3){

hipMemcpy3DParms cpyParms = {0};

cpyParms.dstPtr = make_hipPitchedPtr(*h_data, ((size_t)*m1)*sizeof(Real), ((size_t)*m2), ((size_t)*m3));

cpyParms.srcPtr = make_hipPitchedPtr(*d_data, ((size_t)*md1)*sizeof(Real), ((size_t)*md2), ((size_t)*md3));

cpyParms.extent = make_hipExtent( ((size_t)*m1)*sizeof(Real),  ((size_t)*m3),  ((size_t)*m2));;
cpyParms.kind = hipMemcpyDeviceToDevice;

hipError_t status = hipMemcpy3DAsync(&cpyParms,stream1);

if(status != hipSuccess){fprintf(stderr, "%s\n", hipGetErrorString(status));}

}



// determine which method can be used for allocating data on the GPU
// for now, only valid for 1 MPI process/GPU
extern "C" void FC_FUNC_(cuda_estimate_memory_needs_cu, CUDA_ESTIMATE_MEMORY_NEEDS_CU)(int* iproc, int *N,int *geo, size_t* plansSize, size_t* maxPlanSize, size_t* freeSize, size_t* totalSize) {

 size_t workSize=0;//, maxPlanSize=0, plansSize=0, kernelSize=0, PCGRedSize=0;

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];
 //int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int ysize = NY/2 + geo2 * NY/2;
 int zsize = NZ/2 + geo3 * NZ/2;

//only the first MPI process of the group needs the GPU
if(*iproc==0){
     //size of the cuFFT plans
     // --- Using hipfftEstimate1d
     cufftErrchk(hipfftEstimate1d(NX, HIPFFT_D2Z, ysize*zsize, &workSize));
    // printf("hipfftEstimate1d worksize 1 = %lu\n",workSize);
     *plansSize+=workSize;
     *maxPlanSize=workSize;
     cufftErrchk(hipfftEstimate1d(NX, HIPFFT_Z2D, ysize*zsize, &workSize));
    // printf("hipfftEstimate1d worksize 2 = %lu\n",workSize);
     *plansSize+=workSize;
     *maxPlanSize=std::max(*maxPlanSize,workSize);
     cufftErrchk(hipfftEstimate1d(NY, Transform, (NX/2+1)*zsize, &workSize));
    // printf("hipfftEstimate1d worksize 3 = %lu\n",workSize);
     *plansSize+=workSize;
     *maxPlanSize=std::max(*maxPlanSize,workSize);
     cufftErrchk(hipfftEstimate1d(NZ, Transform, (NX/2+1)*NY, &workSize));
    // printf("hipfftEstimate1d worksize 4 = %lu\n",workSize);
     *plansSize+=workSize;
     *maxPlanSize=std::max(*maxPlanSize,workSize);
    // printf("workSize = %lu\n",plansSize);
}
    // this method could be more precise, but actually seems to answer the same
    //// --- Using hipfftGetSize1d
    //   hipfftHandle plan;
    //   hipfftCreate(&plan);
    //   hipfftGetSize1d(plan, NX, HIPFFT_D2Z, ysize*zsize, &workSize);
    //   printf("hipfftGetSize1d worksize 1 = %lu\n",workSize);
    //   hipfftGetSize1d(plan, NX, HIPFFT_Z2D, ysize*zsize, &workSize);
    //   printf("hipfftGetSize1d worksize 2 = %lu\n",workSize);
    //   hipfftGetSize1d(plan, NY, Transform, (NX/2+1)*zsize, &workSize);
    //   printf("hipfftGetSize1d worksize 3 = %lu\n",workSize);
    //   hipfftGetSize1d(plan, NZ, Transform, (NX/2+1)*NY, &workSize);
    //   printf("hipfftGetSize1d worksize 4 = %lu\n",workSize);

 gpuErrchk(hipMemGetInfo(freeSize,totalSize));

}

// transpose
__global__ void transpose(Complex *idata, Complex *odata,
        int width, int height)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width) {
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
  }
}

// transpose together with spread operation
__global__ void transpose_spread(Complex *idata, Complex *odata, 
	int width, int height, int bign_h)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;
  int div = index_out / bign_h;
  int mod = index_out % bign_h;
  index_out = div * (bign_h << 1) + mod+bign_h;
  int plus = -bign_h;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width) {
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
    #ifdef DOUBLE
      odata[index_out + plus] = make_double2(0., 0.);
    #else
      odata[index_out + plus] = make_float2(0.f, 0.f);
    #endif
  }
}

// transpose together with inverse spread operation
__global__ void transpose_spread_i(Complex *idata, Complex *odata,
        int width, int height, int bign_h)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;
  int div = index_in / bign_h;
  int mod = index_in % bign_h;
  index_in = div * (bign_h << 1) + mod;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width)
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
}

// spread operation
__global__ void spread(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
   unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
   unsigned int tid = threadIdx.x;
 
   Real res = (tid >= spitch) ? src[bid * spitch + tid-spitch] : 0.0;
   if( tid < dpitch) {
	dst[bid * dpitch + tid] = res;
   }
}

// inverse spread operation
__global__ void spread_i(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
   unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
   unsigned int tid = threadIdx.x;

   Real res = src[bid * dpitch + tid];
   if( tid < dpitch) dst[bid * spitch + tid] = res;
}

// spread operation for 2nd dim
__global__ void spread_y(Complex* src, Complex* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Complex res =  src[tid];
   dst[tid1 + blockDim.x*gridDim.x] = res;
#ifdef DOUBLE
   dst[tid1] = make_double2(0., 0.);
#else
   dst[tid1] = make_float2(0.f, 0.f);
#endif
}

__global__ void spread_y_r(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid];
   dst[tid1 + blockDim.x*gridDim.x] = res;
#ifdef DOUBLE
   dst[tid1] = 0.;
#else
   dst[tid1] = 0.f;
#endif
}

__global__ void spread_z(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   //unsigned int tid1 = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid];
   src[tid + (gridDim.y * gridDim.x) * blockDim.x] = res;
#ifdef DOUBLE
   src[tid] = 0.0;
#else
   src[tid] = 0.f;
#endif
}


// inverse spread operation for 2nd dim
__global__ void spread_y_i(Complex* src, Complex* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Complex res =  src[tid1];
   dst[tid] = res;
}

__global__ void spread_y_i_r(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid1];
   dst[tid] = res;
}

// multiply with potential
__global__ void multiply_kernel(int nx, int ny, int nz, Complex *d_data, Real *d_kernel, Real scal) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     d_data[jj + tj+ k*td].x *= d_kernel[jj + tj+ k*td]*scal;
     d_data[jj + tj+ k*td].y *= d_kernel[jj + tj+ k*td]*scal;
 }

}

// scale
__global__ void scale_kernel(int nx, int ny, int nz, Complex *d_data, Real mult) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     d_data[jj + tj+ k*td].x *= mult;
     d_data[jj + tj+ k*td].y *= mult;
 }

}

__global__ void zero(int nx, int ny, int nz, Real *z) {

        int tj = threadIdx.x;
        int td = blockDim.x;

	int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

        int jj = ((blockIdx.y)*gridDim.x + (blockIdx.x))*blockData;

        for (int k=0; k<blockData/td; k++) {
        	z[jj + tj+ k*td] = 0.0;
        }
}

__global__ void copy_0(int nx, int ny, int nz, Real *in, Real *out) {

	int tj = threadIdx.x;
        int td = blockDim.x;

        int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
        int jj1 =  ((blockIdx.y+nz/2)*nx*ny + (blockIdx.x+ny/2)*nx);


        out[jj1+tj+td] = in[jj+tj];

}

__global__ void copy(int nx,int ny,int nz, Real *in, Real *out) {

        int tj = threadIdx.x;
        //int td = blockDim.x;

        int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
        int jj1 =  ((blockIdx.y)*nx*ny + (blockIdx.x)*nx);

        out[jj+tj] = in[jj1+tj];
}

/************ 1D transform *************/

extern "C" void cuda_1d_plan_(int *NX_p, int *Nbatch_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int Nbatch = *Nbatch_p;

 int n1d[3]= {NX, 1, 1};

 cufftErrchk(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, Transform, Nbatch));
 hipfftSetStream(*plan, stream1);
 //hipfftPlan1d(plan, NX, Transform, Nbatch );

}

extern "C" void cuda_1d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   cufftErrchk(TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD));

}

extern "C" void cuda_1d_inverse_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   cufftErrchk( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD));

}

/************ 2D transform *************/

extern "C" void cuda_2d_plan_(int *NX_p, int *NY_p, int *Nbatch_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int NY = *NY_p;
 int Nbatch = *Nbatch_p;

 int n1d[3]= {NX, NY, 1};

 cufftErrchk(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX*NY,
              NULL, 1, NX*NY, Transform, Nbatch));
 hipfftSetStream(*plan, stream1);

}

extern "C" void cuda_2d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {
   cufftErrchk(TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD));
}

extern "C" void cuda_2d_inverse_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {
   cufftErrchk(TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD));
}

/************ 3D transform *************/

extern "C" void cuda_3d_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int n[3] = { NZ, NY, NX };
 cufftErrchk(hipfftPlanMany(plan, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, Transform, 1));
 hipfftSetStream(*plan, stream1);
}

extern "C" void cuda_3d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {
   cufftErrchk(TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD));
}

extern "C" void cuda_3d_inverse_(int *NX_p, int *NY_p, int *NZ_p ,hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   int NX = *NX_p;
   int NY = *NY_p;
   int NZ = *NZ_p;

   cufftErrchk(TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD));

   // scale kernel paramters
   int nThreads = NX;
   dim3 nBlocks(NY,NZ,1);

   scale_kernel <<< nBlocks, nThreads >>> (NX,NY,NZ,*d_data2, 1.0/double(NX*NY*NZ));
}

/************ 3D Poisson Solver for periodic boundary *************/

extern "C" void cuda_3d_psolver_cufft3d_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan, hipfftHandle *plan1) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int n[3] = { NZ, NY, NX };
 cufftErrchk(hipfftPlanMany(plan, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, HIPFFT_D2Z, 1));
 hipfftSetStream(*plan, stream1);

 cufftErrchk(hipfftPlanMany(plan1, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, HIPFFT_Z2D, 1));
 hipfftSetStream(*plan1, stream1);

}


extern "C" void cuda_3d_psolver_cufft3d_(int *NX_p, int *NY_p, int *NZ_p,hipfftHandle *plan,
             hipfftHandle *plan1, Complex **d_data, Complex **d_data2, Real **d_kernel, Real *scal_p,
	     int *geo1_p, int *geo2_p, int *geo3_p) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int geo1 = *geo1_p;
 int geo2 = *geo2_p;
 int geo3 = *geo3_p;

 int ysize = NY/2 + geo2 * NY/2;
 int zsize = NZ/2 + geo3 * NZ/2;

 Real scal = *scal_p;

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NY,NZ,1);

 // copy kernel paramters
 int nthreads = NX/2;
 dim3 nblocks(NY/2,NZ/2,1);

 // spread kernel parameters
 dim3 nblocks_s(zsize,ysize,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

   if (geo1==0 && geo2==0 && geo3==0) {
    src = *d_data;
    dst = *d_data2;
    zero <<< nblocks, nthreads, 0, stream1 >>> (NX,NY,NZ, (Real*)dst);
    copy_0 <<< nblocks, nthreads, 0, stream1  >>> (NX,NY,NZ, (Real*)src, (Real*)dst);
   }
   else {
    if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks_s, NX, 0, stream1>>>((Real*)src, NX/2, (Real*)dst, NX);
    }
    if (geo2==0) {
      if (geo1==0) {
        src = *d_data2;
        dst = *d_data;
      } else {
        src = *d_data;
        dst = *d_data2;
      }
      nblocks_s.x=ysize;
      nblocks_s.y=zsize;
      spread_y_r<<<nblocks_s, NX, 0, stream1>>>((Real*)src, (Real*)dst);
    }
    if (geo3==0) {
      nblocks_s.x=NY;
      nblocks_s.y=zsize;
      spread_z<<<nblocks_s, NX, 0, stream1>>>((Real*)dst, (Real*)src);
    }
   }

   // Forward FFT

   cufftErrchk( hipfftExecD2Z(*plan, (Real*)dst, src));

   // multiply with kernel

   multiply_kernel <<< nBlocks, nThreads, 0, stream1 >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

   // Inverse FFT

   cufftErrchk( hipfftExecZ2D(*plan1, src, (Real*)dst));

   if (geo1==0 && geo2==0 && geo3==0)
     copy <<< nblocks, nthreads, 0, stream1 >>> (NX,NY,NZ, (Real*)dst, (Real*)src);
   else { 
    if (geo2==0) {
       nblocks_s.x=ysize;
       nblocks_s.y=zsize;
       spread_y_i_r<<<nblocks_s, NX, 0, stream1>>>((Real*)dst, (Real*)src);
    }
    if (geo1==0) {
       if (geo2==0) {
        Complex* tmp = src;
        src = dst;
        dst = tmp;
       }
      nblocks_s.x=zsize;
      nblocks_s.y=ysize; 
      spread_i<<<nblocks_s, NX/2, 0, stream1>>>((Real*)dst, NX/2, (Real*)src, NX);
    }
   }
}

/************ 3D Poisson Solver for general boundary *************/

extern "C" void FC_FUNC_(cuda_3d_psolver_general_plan, CUDA_3D_PSOLVER_GENERAL_PLAN)(int *N,
                 hipfftHandle *plan, int *switch_alg,
		 int *geo) {

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 //int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int n1d[3]= {1, 1, 1};

 int ysize = NY/2 + geo2 * NY/2;
 int zsize = NZ/2 + geo3 * NZ/2;

 n1d[0] = NX;
 cufftErrchk(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_D2Z, ysize*zsize));
 hipfftSetStream(*plan, stream1);

 cufftErrchk(hipfftPlanMany(plan+1,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_Z2D, ysize*zsize));
 hipfftSetStream(*(plan+1), stream1);

 n1d[0] = NY;
 cufftErrchk(hipfftPlanMany(plan+2,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize));
 hipfftSetStream(*(plan+2), stream1);

 n1d[0] = NZ;
 cufftErrchk(hipfftPlanMany(plan+3,  1, n1d,
              NULL, 1, NZ,
              NULL, 1, NZ, Transform, (NX/2+1)*NY));
 hipfftSetStream(*(plan+3), stream1);

 *switch_alg = 0;

 /*int nPrimeSize = 17;
 int primeSize[] = {92,104,116,124,136,148,152,164,172,184,188,204,208,220,228,232,248};

 for (int p=0; p<nPrimeSize; p++)
   if (NZ == primeSize[p]) {
     *switch_alg = 1;
     break;
   }

 n1d[0] = NZ;

 int inembed[1];
 int onembed[1];
 inembed[0] = 1;
 onembed[0] = 1;
 if(hipfftPlanMany(plan+4,  1, n1d,
              inembed, NY, 1,
              onembed, NY, 1, Transform, NY) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");*/

}

extern "C" void FC_FUNC_(cuda_3d_psolver_general, CUDA_3D_PSOLVER_GENERAL)(int *N,
          hipfftHandle *plan,
          Complex **d_data, Complex **d_data2, Real **d_kernel, int *switch_alg,
          int *geo, Real *scal_p) {

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 Real scal = *scal_p;

 int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int ysize=NY/2+geo2*NY/2;
 int zsize=NZ/2+geo3*NZ/2;

 // transpose kernel parameters
 dim3 grid((NX/2+1+TILE_DIM-1)/TILE_DIM,(ysize*zsize+TILE_DIM-1)/TILE_DIM,1);
 dim3 threads(TILE_DIM,TILE_DIM,1);

 // spread kernel parameters
 dim3 nblocks(zsize,ysize,1);

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NZ,NY,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

 // X transform 

   if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks, NX, 0, stream1>>>((Real*)src, NX/2, (Real*)dst, NX);
   }

   cufftErrchk(hipfftExecD2Z(plan[0], (Real*)dst, src));

   if (geo2==0) {
     transpose_spread <<< grid, threads, 0, stream1 >>>(src, dst,NX/2+1,ysize*zsize,NY/2);
   } else {
     transpose <<< grid, threads, 0, stream1 >>>(src, dst,NX/2+1,ysize*zsize);
   }

   // Y transform
   cufftErrchk(TransformExec(plan[2], dst, src, HIPFFT_FORWARD));

  // Z transform, on entire cube
  if (!(*switch_alg)) {
   grid.x = (NY+TILE_DIM-1)/TILE_DIM;
   grid.y = ((NX/2+1)*zsize+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread <<< grid, threads, 0, stream1 >>>(src,dst,NY,(NX/2+1)*NZ/2,NZ/2);
   } else {
     transpose <<< grid, threads, 0, stream1 >>>(src, dst,NY,(NX/2+1)*NZ);
   }

   cufftErrchk(TransformExec(plan[3], dst, src, HIPFFT_FORWARD));
  }
  else {
   if (geo3==0) {
      nblocks.x=zsize;
      nblocks.y=NX;
      spread_y<<<nblocks, NY, 0, stream1>>>(src, dst);
   }

   for(int k=0; k<NX; ++k){
     cufftErrchk(TransformExec(plan[4], dst, src, HIPFFT_FORWARD));
     src += NY*NZ;
     dst += NY*NZ;
   }

   src -= NX*NY*NZ;
   dst -= NX*NY*NZ;
  }

  // multiply with kernel

  multiply_kernel <<< nBlocks, nThreads, 0, stream1>>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

  // inverse transform

  // Z transform, on entire cube 
  if (!(*switch_alg)) {
   cufftErrchk(TransformExec(plan[3], src, dst, HIPFFT_BACKWARD));

   grid.x = (zsize*(NX/2+1)+TILE_DIM-1)/TILE_DIM;
   grid.y = (NY+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread_i <<< grid, threads, 0, stream1 >>>(dst, src,NZ/2*(NX/2+1),NY,NZ/2);
   } else {
     transpose <<< grid, threads, 0, stream1 >>>(dst, src,NZ*(NX/2+1),NY);
   }

  }
  else {

   for(int k=0; k<NX; ++k){
     cufftErrchk(TransformExec(plan[4], src, dst, HIPFFT_BACKWARD));
     src += NY*NZ;
     dst += NY*NZ;
   }

   src -= NX*NY*NZ;
   dst -= NX*NY*NZ;

   if (geo3==0)
      spread_y_i<<<nblocks, NY, 0, stream1>>>(dst, src);
  }

  // Y transform

   cufftErrchk(TransformExec(plan[2], src, dst, HIPFFT_BACKWARD));

   grid.x = (ysize*zsize+TILE_DIM-1)/TILE_DIM;
   grid.y = (NX/2+1+TILE_DIM-1)/TILE_DIM;

   if (geo2==0) {
      transpose_spread_i <<< grid, threads, 0, stream1 >>>(dst, src,ysize*zsize,NX/2+1, NY/2);
   } else
      transpose <<< grid, threads, 0, stream1 >>>(dst, src,ysize*zsize,NX/2+1);

   // X transform

   cufftErrchk(hipfftExecZ2D(plan[1], src, (Real*)dst));

   nblocks.x=zsize;
   nblocks.y=ysize;
   if (geo1==0) {
      spread_i<<<nblocks, NX/2, 0, stream1>>>((Real*)dst,NX/2, (Real*)src, NX);
   }
}


extern "C" void FC_FUNC_(cuda_3d_psolver_plangeneral, CUDA_3D_PSOLVER_PLANGENERAL)(int *N,
          Complex **d_data, Complex **d_data2, Real **d_kernel,
          int *geo, Real *scal_p) {

 hipfftHandle plan;

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 Real scal = *scal_p;

 int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int ysize=NY/2+geo2*NY/2;
 int zsize=NZ/2+geo3*NZ/2;

 // transpose kernel parameters
 dim3 grid((NX/2+1+TILE_DIM-1)/TILE_DIM,(ysize*zsize+TILE_DIM-1)/TILE_DIM,1);
 dim3 threads(TILE_DIM,TILE_DIM,1);

 // spread kernel parameters
 dim3 nblocks(zsize,ysize,1);

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NZ,NY,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

 int n1d[3]= {1, 1, 1};

 n1d[0] = NX;
 cufftErrchk(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_D2Z, ysize*zsize));
 hipfftSetStream(plan, stream1);

 // X transform 

   if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks, NX, 0, stream1>>>((Real*)src, NX/2, (Real*)dst, NX);
   }

   cufftErrchk(hipfftExecD2Z(plan, (Real*)dst, src));

   if (geo2==0) {
     transpose_spread <<< grid, threads, 0, stream1 >>>(src, dst,NX/2+1,ysize*zsize,NY/2);
   } else {
     transpose <<< grid, threads, 0, stream1 >>>(src, dst,NX/2+1,ysize*zsize);
   }

   hipfftDestroy(plan);

   n1d[0] = NY;
   cufftErrchk(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize));
   hipfftSetStream(plan, stream1);

   // Y transform
   cufftErrchk(TransformExec(plan, dst, src, HIPFFT_FORWARD));

  // Z transform, on entire cube
   grid.x = (NY+TILE_DIM-1)/TILE_DIM;
   grid.y = ((NX/2+1)*zsize+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread <<< grid, threads, 0, stream1 >>>(src, dst,NY,(NX/2+1)*NZ/2,NZ/2);
   } else {
     transpose <<< grid, threads, 0, stream1 >>>(src, dst,NY,(NX/2+1)*NZ);
   }

   hipfftDestroy(plan);
   n1d[0] = NZ;
   cufftErrchk(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NZ,
              NULL, 1, NZ, Transform, (NX/2+1)*NY));
   hipfftSetStream(plan, stream1);

   cufftErrchk(TransformExec(plan, dst, src, HIPFFT_FORWARD));

  // multiply with kernel

  multiply_kernel <<< nBlocks, nThreads, 0, stream1 >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

  // inverse transform

  // Z transform, on entire cube 
   cufftErrchk(TransformExec(plan, src, dst, HIPFFT_BACKWARD));

   grid.x = (zsize*(NX/2+1)+TILE_DIM-1)/TILE_DIM;
   grid.y = (NY+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread_i <<< grid, threads, 0, stream1 >>>(dst, src,NZ/2*(NX/2+1),NY,NZ/2);
   } else {
     transpose <<< grid, threads, 0, stream1 >>>(dst, src,NZ*(NX/2+1),NY);
   }

  // Y transform

   hipfftDestroy(plan);
   n1d[0] = NY;
   cufftErrchk(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize));
   hipfftSetStream(plan, stream1);

   cufftErrchk(TransformExec(plan, src, dst,HIPFFT_BACKWARD));

   grid.x = (ysize*zsize+TILE_DIM-1)/TILE_DIM;
   grid.y = (NX/2+1+TILE_DIM-1)/TILE_DIM;

   if (geo2==0) {
      transpose_spread_i <<< grid, threads, 0, stream1 >>>(dst,src,ysize*zsize,NX/2+1, NY/2);
   } else
      transpose <<< grid, threads, 0, stream1 >>>(dst, src,ysize*zsize,NX/2+1);

   // X transform

   hipfftDestroy(plan);
   n1d[0] = NX;
   cufftErrchk(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_Z2D, ysize*zsize));
   hipfftSetStream(plan, stream1);

   cufftErrchk(hipfftExecZ2D(plan, src, (Real*)dst));

   nblocks.x=zsize;
   nblocks.y=ysize;
   if (geo1==0) {
      spread_i<<<nblocks, NX/2, 0, stream1>>>((Real*)dst,NX/2, (Real*)src, NX);
   }

   hipfftDestroy(plan);
}


//Specialization of the computation part for each reduction kernel.
//the kern1_red itself is useless as it is the same for all 3 reductions
//keeping it, as we may want to use another someday

typedef void(*comp_and_red_op)(int, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*);
typedef void(*red_op)(int, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*);

__device__
void kern1_comp_and_red (int i , Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata, Real* sum){
  Real zeta=z_GPU[i]*oneoeps_GPU[i];
  z_GPU[i]=zeta;
  *sum+= (r_GPU[i]*zeta);
}

__device__
void kern1_red (int i , Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata, Real* sum){
  *sum+= (g_odata[i]);
}


__device__
void kern2_comp_and_red (int i , Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata, Real* sum){
  Real zeta=z_GPU[i];
  Real pval = zeta+(*beta_GPU / *beta0_GPU)*p_GPU[i];
  Real qval = zeta*corr_GPU[i]+r_GPU[i]+(*beta_GPU / *beta0_GPU)*q_GPU[i];
  p_GPU[i] = pval;
  q_GPU[i] = qval;
  *sum+= (pval*qval);
}

__device__
void kern3_comp_and_red (int i , Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata, Real* sum){
  x_GPU[i] = x_GPU[i] + *alpha_GPU*p_GPU[i];
  r_GPU[i] = r_GPU[i] - *alpha_GPU*q_GPU[i];
  z_GPU[i] = r_GPU[i] * oneoeps_GPU[i];
  *sum+=(r_GPU[i]*r_GPU[i]);
}

__device__
void kern_finalize_and_red (int i , Real* zf_GPU, Real* rho_GPU, Real* , Real* , Real* , Real* , Real* , Real* , Real* , Real* , Real*, Real* , Real* sum){
  Real pt =zf_GPU[i];
  *sum+= (rho_GPU[i]*pt);
  rho_GPU[i]=pt;
}

__device__
void kern_finalize_and_red_sumpion (int i , Real* zf_GPU, Real* rho_GPU, Real* pot_ionGPU, Real* , Real* , Real* , Real* , Real* , Real* , Real* , Real*, Real* , Real* sum){
  Real pt =zf_GPU[i];
  *sum+= (rho_GPU[i]*pt);
  rho_GPU[i]=pt+pot_ionGPU[i];
}


//helper functions for the reduction (reduction taken from NVIDIA cuda samples)
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};
// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};



/*actual kernel call for the reduction, that is specialized with 2 template 
subkernels, one for computation, the other for specific reduction part.
Result is written in g_odata array in GPU memory. So this must be called several 
times to actually reduce to a single element.
*/
template <unsigned int blockSize, bool nIsPow2, comp_and_red_op op1, red_op op2>
__global__ void
reduce_kernel(int n, int reduceOnly, Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata)
{
    Real *sdata = SharedMemory<Real>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    Real mySum = 0;
    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
    if(!reduceOnly){
        op1 (i , p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, g_odata, &mySum);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n){
            op1 (i + blockSize, p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, g_odata, &mySum);
        }
    }else{

        //subsequent calls to the kernel after the first one don't have to perform 
        // the computations
        op2 (i, p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, g_odata, &mySum);

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            op2 (i + blockSize, p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, g_odata, &mySum);
    }

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}


//wrapper for templated kernel
template <comp_and_red_op op1, red_op op2>
void reduce_step(int s, int threads, int blocks, int reduceOnly,  Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* d_odata){
    //TODO : 2D
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(Real) : threads * sizeof(Real);

    if (((s&(s-1))==0))//pow2
    {
        switch (threads)
        {
            case 512:
                reduce_kernel<512, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 256:
                reduce_kernel<256, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;

            case 128:
                reduce_kernel<128, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 64:
                reduce_kernel<64, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 32:
                reduce_kernel<32, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 16:
                reduce_kernel<16, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  8:
                reduce_kernel<8, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  4:
                reduce_kernel<4, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  2:
                reduce_kernel<2, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  1:
                reduce_kernel<1, true, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
        }
    }
    else
    {
        switch (threads)
        {
            case 512:
                reduce_kernel<512, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 256:
                reduce_kernel<256, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 128:
                reduce_kernel<128, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 64:
                reduce_kernel<64, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 32:
                reduce_kernel<32, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 16:
                reduce_kernel<16, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  8:
                reduce_kernel<8, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  4:
                reduce_kernel<4, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  2:
                reduce_kernel<2, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  1:
                reduce_kernel<1, false, op1, op2><<< dimGrid, dimBlock, smemSize, stream1 >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
        }
    }

//gpuErrchk( hipPeekAtLastError() );
//gpuErrchk( hipDeviceSynchronize() );

}

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

/*this performs some calculations to chose the size of the blocks we want to use 
for reduction, while limiting their number for efficiency purposes, as each kernel
 will handle several elements in this version (see last version of reduction in
reduction sample code from Nvidia)
*/
template <comp_and_red_op op1, red_op op2>
void apply_reduction(int n,
          Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU,Real* d_odata, Real* result, int retrieve) {
    int maxThreads=256;
    int maxBlocks=64;
    int blocks=0;
    int threads=0;

    //get device capability, to avoid block/grid size excceed the upbound
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
    blocks = (n + (threads * 2 - 1)) / (threads * 2);

    if ((Real)threads*blocks > (Real)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        printf("n is too large, please choose a smaller number!\n");
    }

    if (blocks > prop.maxGridSize[0])
    {
        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
               blocks, prop.maxGridSize[0], threads*2, threads);

        blocks /= 2;
        threads *= 2;
    }

    //we will only use maxblocks blocks, and make each thread work on more data
    blocks = min(maxBlocks, blocks);


//    Real *d_odata = NULL;
//    hipMalloc((void **) &d_odata, blocks*sizeof(Real));
//    gpuErrchk( hipPeekAtLastError() );
    //first reduction
    //hipDeviceSynchronize();
    reduce_step<op1, op2>(n, threads, blocks, 0,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
  
    gpuErrchk( hipPeekAtLastError() );

    int s=blocks;
    //loop and perform as many reductions steps as necessary
    while (s > 1)
    {
        threads = (s < maxThreads*2) ? nextPow2((s + 1)/ 2) : maxThreads;
        blocks = (s + (threads * 2 - 1)) / (threads * 2);
        if (blocks > prop.maxGridSize[0])
        {
            printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
            blocks, prop.maxGridSize[0], threads*2, threads);

            blocks /= 2;
            threads *= 2;
        }
        blocks = min(maxBlocks, blocks);

        reduce_step<op1, op2>(s, threads, blocks, 1, p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
        gpuErrchk( hipPeekAtLastError() );
        s = (s + (threads*2-1)) / (threads*2);
    }
  if(retrieve != 0){
    hipMemcpyAsync(result, d_odata, sizeof(Real), hipMemcpyDeviceToHost,stream1);
  }else{
  gpuErrchk( hipPeekAtLastError() );
    //for this one the value will be kept on the card, accumulation will be performed later
    hipMemcpyAsync(*(Real**)result, d_odata, sizeof(Real), hipMemcpyDeviceToDevice,stream1);
  }
  gpuErrchk( hipPeekAtLastError() );
//  hipFree(d_odata);
}

//these will be called from fortran, and apply the reduction with the right subkernels

extern "C" void FC_FUNC_(first_reduction_kernel, FIRST_REDUCTION_KERNEL)(int* n1, int* n23,
          Real** p_GPU, Real** q_GPU, Real** r_GPU, Real** x_GPU, Real** z_GPU, Real** corr_GPU, Real** oneoeps_GPU, Real** alpha_GPU, Real** beta_GPU, Real** beta0_GPU, Real** kappa_GPU, Real** d_odata, Real* result) {

    int n=(*n1) * (*n23);
    apply_reduction<kern1_comp_and_red, kern1_red>(n, *p_GPU, *q_GPU, *r_GPU, *x_GPU, *z_GPU, *corr_GPU, *oneoeps_GPU, *alpha_GPU, *beta_GPU, *beta0_GPU, *kappa_GPU, *d_odata, result,1);

}

extern "C" void FC_FUNC_(second_reduction_kernel, SECOND_REDUCTION_KERNEL)(int* n1, int* n23,
          Real** p_GPU, Real** q_GPU, Real** r_GPU, Real** x_GPU, Real** z_GPU, Real** corr_GPU, Real** oneoeps_GPU, Real** alpha_GPU, Real** beta_GPU, Real** beta0_GPU, Real** kappa_GPU, Real** d_odata, Real* result) {

    int n=(*n1) * (*n23);
    apply_reduction<kern2_comp_and_red, kern1_red>(n, *p_GPU, *q_GPU, *r_GPU, *x_GPU, *z_GPU, *corr_GPU, *oneoeps_GPU, *alpha_GPU, *beta_GPU, *beta0_GPU, *kappa_GPU, *d_odata, result,1);

}

extern "C" void FC_FUNC_(third_reduction_kernel, THIRD_REDUCTION_KERNEL)(int* n1, int* n23,
          Real** p_GPU, Real** q_GPU, Real** r_GPU, Real** x_GPU, Real** z_GPU, Real** corr_GPU, Real** oneoeps_GPU, Real** alpha_GPU, Real** beta_GPU, Real** beta0_GPU, Real** kappa_GPU, Real** d_odata, Real* result) {

    int n=(*n1) * (*n23);
    apply_reduction<kern3_comp_and_red, kern1_red>(n, *p_GPU, *q_GPU, *r_GPU, *x_GPU, *z_GPU, *corr_GPU, *oneoeps_GPU, *alpha_GPU, *beta_GPU, *beta0_GPU, *kappa_GPU, *d_odata, result,1);

}

extern "C" void FC_FUNC_(finalize_reduction_kernel, THIRD_REDUCTION_KERNEL)(int* sumpion, int* n1, int* n23,int* m1, int* m23,
          Real** zf_GPU, Real** rho_GPU, Real** pot_ion_GPU, Real** d_odata, Real* result,int* retrieve) {

    int n=(*n1) * (*n23);
if(!*sumpion)
    apply_reduction<kern_finalize_and_red, kern1_red>(n, *zf_GPU, *rho_GPU, NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL, *d_odata, result,*retrieve);
else
    apply_reduction<kern_finalize_and_red_sumpion, kern1_red>(n, *zf_GPU, *rho_GPU, *pot_ion_GPU, NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL, *d_odata, result,*retrieve);

}


__global__ void pre_computation_kernel(int nx, int ny, int nz,  Real *rho, Real *data1, int shift1,Real *data2,int shift2, Real hfac) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     int idx =jj + tj+ k*td;
     rho[idx] =  hfac*data1[idx+shift1]*data2[idx+shift2];
 }

}

extern "C" void FC_FUNC_(gpu_pre_computation,GPU_PRE_COMPUTATION)(int* NX_p, int* NY_p, int* NZ_p, Real** rho_GPU, Real** data1_GPU, int* shift1, Real** data2_GPU, int* shift2, Real* hfac){
//    !$omp parallel do default(shared) private(i)
//    do i=1,ndim
//      rp_ij(i)=hfac*phi1%data(i+shift1)*phi2%data(i+shift2)
//    end do
//    !$omp end parallel do

   int NX = *NX_p;
   int NY = *NY_p;
   int NZ = *NZ_p;

   // scale kernel paramters
   int nThreads = NX;
   dim3 nBlocks(NY,NZ,1);
   pre_computation_kernel <<< nBlocks, nThreads, 0, stream1 >>> (NX,NY,NZ,*rho_GPU, *data1_GPU,*shift1,*data2_GPU,*shift2,*hfac);
 // hipDeviceSynchronize();
 // gpuErrchk( hipPeekAtLastError() );
}

__global__ void post_computation_kernel(int nx, int ny, int nz,  Real *rho, Real *data1, int shift1,Real *data2,int shift2, Real hfac) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     int idx =jj + tj+ k*td;
     data1[idx+shift1] = data1[idx+shift1] + hfac*rho[idx]*data2[idx+shift2];
 }

}

extern "C" void FC_FUNC_(gpu_post_computation,GPU_POST_COMPUTATION)(int* NX_p, int* NY_p, int* NZ_p, Real** rho_GPU, Real** data1_GPU, int* shift1, Real** data2_GPU, int* shift2, Real* hfac){
//  do i=1,ndim
//    phi1%res(i+shift1_res)=phi1%res(i+shift1_res)+hfac1*rp_ij(i)*phi2%data(i+shift2)
//  end do
   int NX = *NX_p;
   int NY = *NY_p;
   int NZ = *NZ_p;

   // scale kernel paramters
   int nThreads = NX;
   dim3 nBlocks(NY,NZ,1);

   post_computation_kernel <<< nBlocks, nThreads, 0, stream1 >>> (NX,NY,NZ,*rho_GPU, *data1_GPU,*shift1,*data2_GPU,*shift2,*hfac);

//  hipDeviceSynchronize();
 // gpuErrchk( hipPeekAtLastError() );
}

__global__ void accumulate_eexctX_kernel(Real* ehart_GPU, Real* eexctX_GPU, Real hfac) {

 if(threadIdx.x==0){
    *eexctX_GPU=*eexctX_GPU+*ehart_GPU*hfac;
  };
}


extern "C" void FC_FUNC_(gpu_accumulate_eexctx,GPU_ACCUMULATE_EEXCTX)(Real** ehart_GPU, Real** eexctX_GPU, Real* hfac){

   accumulate_eexctX_kernel <<< 1, 1, 0, stream1 >>> (*ehart_GPU, *eexctX_GPU,*hfac);

//  hipDeviceSynchronize();
 // gpuErrchk( hipPeekAtLastError() );
}

